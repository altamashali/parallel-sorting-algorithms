#include "hip/hip_runtime.h"
#include <stdio.h>

#include <time.h>
#include <stdlib.h>

#define SIZE 8192
#define THREADSIZE 64
#define BLOCKSIZE ((SIZE-1)/THREADSIZE + 1) 
#define RADIX 10

__global__ void copyKernel(int * inArray, int * semiSortArray, int arrayLength){

    int index   = blockIdx.x * blockDim.x + threadIdx.x;

    if(index < arrayLength){
        inArray[index]      = semiSortArray[index];
    }
}

__global__ void histogramKernel(int * inArray, int * outArray, int * radixArray, int arrayLength, int significantDigit){

    __shared__ int inArrayShared[THREADSIZE];
    __shared__ int outArrayShared[RADIX];
    __shared__ int radixArrayShared[THREADSIZE];

    int index   = blockIdx.x * blockDim.x + threadIdx.x;
    int thread  = threadIdx.x;
    int blockIndex  = blockIdx.x * RADIX;

    int radix;
    int arrayElement;
    int i;

    if(thread ==  0){
        for(i =0; i < RADIX; i ++){
            outArrayShared[i]       = 0;
        }
    }

    if(index < arrayLength){
        inArrayShared[thread]       = inArray[index];
    }

    __syncthreads();

    if(index < arrayLength)
    {   
        arrayElement            = inArrayShared[thread];
        radix               = ((arrayElement/significantDigit) % 10);
        radixArrayShared[thread]    = radix;

        atomicAdd(&outArrayShared[radix], 1);
    }

    if(index < arrayLength){
        radixArray[index]       = radixArrayShared[thread];
    }

    if(thread == 0){
        for(i =0; i < RADIX; i ++){
            outArray[blockIndex + i]        = outArrayShared[i];
        }
    }
}

__global__ void combineBucket(int * blockBucketArray, int * bucketArray){

    __shared__ int bucketArrayShared[RADIX];

    int index   = blockIdx.x * blockDim.x + threadIdx.x;

    int i;

    bucketArrayShared[index]    = 0;

    for(i = index; i < RADIX*BLOCKSIZE; i=i+RADIX){
        atomicAdd(&bucketArrayShared[index], blockBucketArray[i]);      
    } 

    bucketArray[index]      = bucketArrayShared[index];
}


__global__ void indexArrayKernel(int * radixArray,  int * bucketArray, int * indexArray, int arrayLength, int significantDigit){

    int index   = blockIdx.x * blockDim.x + threadIdx.x;

    int i;
    int radix;
    int pocket;

    if(index < RADIX){

        for(i = 0; i < arrayLength; i++){

            radix           = radixArray[arrayLength -i -1];
            if(radix == index){
                pocket              = --bucketArray[radix];
                indexArray[arrayLength -i -1]   = pocket;       
            }
        }
    }
}

__global__ void semiSortKernel(int * inArray, int * outArray, int* indexArray, int arrayLength, int significantDigit){

    int index   = blockIdx.x * blockDim.x + threadIdx.x;

    int arrayElement;
    int arrayIndex;

    if(index < arrayLength){
        arrayElement            = inArray[index];
        arrayIndex          = indexArray[index];
        outArray[arrayIndex]        = arrayElement;
    }



}

void printArray(int * array, int size){
    int i;
    printf("[ ");
    for (i = 0; i < size; i++)
        printf("%d ", array[i]);
    printf("]\n");
}

int findLargestNum(int * array, int size){
    int i;
    int largestNum = -1;
    for(i = 0; i < size; i++){
        if(array[i] > largestNum)
            largestNum = array[i];
    }
    return largestNum;
}


void cudaScanThrust(int* inarray, int arr_length, int* resultarray) {

        int length = arr_length;

    thrust::device_ptr<int> d_input = thrust::device_malloc<int>(length);
        thrust::device_ptr<int> d_output = thrust::device_malloc<int>(length);

        hipMemcpy(d_input.get(), inarray, length * sizeof(int), hipMemcpyHostToDevice);

        thrust::inclusive_scan(d_input, d_input + length, d_output);

        hipDeviceSynchronize();

        hipMemcpy(resultarray, d_output.get(), length * sizeof(int), hipMemcpyDeviceToHost);

        thrust::device_free(d_input);
        thrust::device_free(d_output);
}

void radixSort(int * array, int size){

    double startTime;
    double endTime;
    double duration;

    int significantDigit    = 1;

    int threadCount;
    int blockCount;

    threadCount             = THREADSIZE;
    blockCount          = BLOCKSIZE;;

    int * outputArray;
    int * inputArray;
    int * radixArray;
    int * bucketArray;
    int * indexArray;
    int * semiSortArray;
    int * blockBucketArray;

    hipMalloc((void **)& inputArray, sizeof(int)*size);
    hipMalloc((void **)& indexArray, sizeof(int)*size);
    hipMalloc((void **)& radixArray, sizeof(int)*size);
    hipMalloc((void **)& outputArray, sizeof(int)*size);
    hipMalloc((void **)& semiSortArray, sizeof(int)*size);
    hipMalloc((void **)& bucketArray, sizeof(int)*RADIX);
    hipMalloc((void **)& blockBucketArray, sizeof(int)*RADIX*BLOCKSIZE);   


    hipMemcpy(inputArray, array, sizeof(int)*size, hipMemcpyHostToDevice);

    int largestNum;
    thrust::device_ptr<int>d_in     = thrust::device_pointer_cast(inputArray);
    thrust::device_ptr<int>d_out;
    d_out = thrust::max_element(d_in, d_in + size);
    largestNum      = *d_out;   
    printf("\tLargestNumThrust : %d\n", largestNum);

    startTime   = CycleTimer::currentSeconds(); 

    while (largestNum / significantDigit > 0){

        int bucket[RADIX] = { 0 };
        hipMemcpy(bucketArray, bucket, sizeof(int)*RADIX, hipMemcpyHostToDevice);

        histogramKernel<<<blockCount, threadCount>>>(inputArray, blockBucketArray, radixArray, size, significantDigit);     
        hipDeviceSynchronize();

        combineBucket<<<1, RADIX>>>(blockBucketArray,bucketArray);
        hipDeviceSynchronize();            

        cudaScanThrust(bucketArray, RADIX, bucketArray);    
        hipDeviceSynchronize();

        indexArrayKernel<<<blockCount, threadCount>>>(radixArray, bucketArray, indexArray, size, significantDigit);
        hipDeviceSynchronize();

        semiSortKernel<<<blockCount, threadCount>>>(inputArray, semiSortArray, indexArray, size, significantDigit);
        hipDeviceSynchronize();

        copyKernel<<<blockCount, threadCount>>>(inputArray, semiSortArray, size);
        hipDeviceSynchronize();


        significantDigit *= RADIX;

    }

    endTime     = CycleTimer::currentSeconds();
    duration    = endTime - startTime;

    hipMemcpy(array, semiSortArray, sizeof(int)*size, hipMemcpyDeviceToHost);

    printf("Duration : %.3f ms\n", 1000.f * duration);

    hipFree(inputArray);
    hipFree(indexArray);
    hipFree(radixArray);
    hipFree(bucketArray);
    hipFree(blockBucketArray);
    hipFree(outputArray);
    hipFree(semiSortArray);
}

int main(){

    printf("\n\nRunning Radix Sort Example in C!\n");
    printf("----------------------------------\n");

    int size = SIZE;
    int* array;
    int i;
    int list;

    srand(time(NULL));

    for(i =0; i < size; i++){
        list[i]     = SIZE -i;
    }

    array = &list[0];
    printf("\nUnsorted List: ");
    printArray(array, size);

    radixSort(array, size);

    printf("\nSorted List:");
    printArray(array, size);

    printf("\n");

    return 0;